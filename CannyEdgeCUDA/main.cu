#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <opencv2/opencv.hpp>
#include <sstream>
#include <string>

#include "kernels.h"

#define checkCudaError() _checkCudaError(__FILE__, __LINE__, __func__)

using namespace cv;
using namespace std;

bool headless = false;
int lowThreshold = 20, // %
highThreshold = 60;
string media = "image"; // or video

dim3 blockSize = dim3(16, 16, 1); // thread dimensions of each block

float* devGaussKernel; // Gaussian convolution kernel pointer

inline void _checkCudaError(const char* file, int line, const char* function) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error at %s:%d, in function %s: %s\n",
            file, line, function, hipGetErrorString(err));
        exit(1);
    }
}

void canny(Mat& img) {
    hipEvent_t start, stop;
    if (media != "video") {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
    }

    int width = img.cols,
        height = img.rows;
    size_t rgbDataSize = img.total() * img.channels(),
        grayDataSize = img.total(); // One channel, for grayscale

    // Grayscale
    unsigned char* devRgbData;
    hipMalloc(&devRgbData, rgbDataSize);
    hipMemcpy(devRgbData, img.data, rgbDataSize, hipMemcpyHostToDevice);

    unsigned char* devGrayData;
    hipMalloc(&devGrayData, grayDataSize);

    dim3 gridSize = dim3((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, 1);

    grayscale << <gridSize, blockSize >> > (devRgbData, devGrayData, width, height);
    hipFree(devRgbData);

    // Gaussian blur
    unsigned char* devGrayDataCopy;
    hipMalloc(&devGrayDataCopy, grayDataSize);
    hipMemcpy(devGrayDataCopy, devGrayData, grayDataSize, hipMemcpyDeviceToDevice);

    gaussianBlur << <gridSize, blockSize >> > (devGaussKernel, devGrayDataCopy, devGrayData, width, height);
    hipFree(devGrayDataCopy);

    // Intensity gradient
    float* devDirections, * devMagnitudes;
    hipMalloc(&devMagnitudes, grayDataSize * sizeof(float));
    hipMalloc(&devDirections, grayDataSize * sizeof(float));
    intensityGradient << <gridSize, blockSize >> > (devGrayData, width, height, devMagnitudes, devDirections);

    nonMaximumSuppression << <gridSize, blockSize >> > (devGrayData,
        width,
        height,
        devDirections,
        devMagnitudes,
        lowThreshold / 100.0f * 255.0f,
        highThreshold / 100.0f * 255.0f);
    checkCudaError();
    hipFree(devDirections);
    hipFree(devMagnitudes);

    // Hysteresis and copy to host
    hysteresis << <gridSize, blockSize >> > (devGrayData, width, height);

    unsigned char* hostImgData;
    hipHostAlloc((void**)&hostImgData, grayDataSize, hipHostMallocDefault); // Pinned memory
    hipMemcpy(hostImgData, devGrayData, grayDataSize, hipMemcpyDeviceToHost);

    hipFree(devGrayData);

    // Show or write to a file
    Mat modifiedImg = Mat(height, width, CV_8UC1, hostImgData);
    if (headless) {
        if (media == "image") {
            imwrite("output.bmp", modifiedImg);
            printf("Image saved as output.bmp\n");
        }
        else {
            // Convert and copy image data, needed for video output
            cvtColor(modifiedImg, img, COLOR_GRAY2BGR);
        }
    }
    else {
        imshow("Canny edge detection", modifiedImg);
    }

    hipHostFree(hostImgData);

    if (media != "video") {
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        printf("Canny took %.2fms\n", elapsed);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
}

// Common for all kernel executions
void initGauss() {
    float kernel[5][5]{},
        sum = .0f,
        sigma = .75f;

    for (int x = -2;x <= 2;x++) {
        for (int y = -2;y <= 2;y++) {
            int i = x + 2,
                j = y + 2;
            kernel[i][j] = (float)(1 / (2 * M_PI * sigma * sigma) * exp(-(x * x + y * y) / 2.0f * sigma * sigma));
            sum += kernel[i][j];
        }
    }

    for (int i = 0;i < 5;i++) {
        for (int j = 0;j < 5;j++) {
            kernel[i][j] /= sum; // Normalize
        }
    }

    printf("Gaussian blur kernel:\n");
    for (int i = 0;i < 5;i++) {
        for (int j = 0;j < 5;j++) {
            printf("%.2f ", kernel[i][j]);
        }
        printf("\n");
    }

    int kernelDataSize = 5 * 5 * sizeof(float);
    hipMalloc(&devGaussKernel, kernelDataSize);
    hipMemcpy(devGaussKernel, (float*)kernel, kernelDataSize, hipMemcpyHostToDevice);
}

// sliding
void onTrackbar(int, void* userdata) {
    if (lowThreshold > highThreshold) {
        printf("Let's be reasonable here\n");
        highThreshold = lowThreshold;
        setTrackbarPos("High (%)", "Canny edge detection", highThreshold);
        return;
    }

    printf("Low threshold: %d, high: %d\n", lowThreshold, highThreshold);

    if (media == "image") {
        Mat* img = (Mat*)userdata;
        canny(*img);
    }
}

void handleImage(string& inPath) {
    Mat img = imread(inPath, IMREAD_ANYCOLOR);

    if (img.empty()) {
        printf("Could not open the image: %s\n", inPath.c_str());
        return;
    }

    if (headless) {
        canny(img);
    }
    else {
        namedWindow("Original", WINDOW_AUTOSIZE);
        imshow("Original", img);

        namedWindow("Canny edge detection", WINDOW_AUTOSIZE);
        createTrackbar("Low (%)", "Canny edge detection", &lowThreshold, 100, onTrackbar, &img);
        createTrackbar("High (%)", "Canny edge detection", &highThreshold, 100, onTrackbar, &img);

        onTrackbar(0, &img); // First render

        printf("Press q to quit\n");
        while (true) {
            if ((char)waitKey(30) == 'q') break;
        }
    }
}

void handleVideo(string& inPath) {
    VideoCapture cap(inPath);
    if (!cap.isOpened()) {
        printf("Could not open the video: %s\n", inPath.c_str());
        return;
    }
    printf("Video opened successfully\n");

    Mat frame;

    if (headless) {
        int width = (int)cap.get(CAP_PROP_FRAME_WIDTH),
            height = (int)cap.get(CAP_PROP_FRAME_HEIGHT),
            fps = (int)cap.get(cv::CAP_PROP_FPS);
        printf("%dx%d %dfps\n", width, height, fps);

        VideoWriter output("output.avi", VideoWriter::fourcc('M', 'J', 'P', 'G'), fps, Size(width, height));
        if (!output.isOpened()) {
            printf("Could not open video for writing: output.avi\n");
            return;
        }

        int frameId = 0;
        while (cap.read(frame)) {
            if (++frameId % 100 == 0) {
                printf("Processing frame %d\n", frameId);
            }
            canny(frame);
            output.write(frame);
        }

        output.release();
    }
    else {
        printf("Press q to quit\n");

        namedWindow("Canny edge detection", WINDOW_AUTOSIZE);
        createTrackbar("Low (%)", "Canny edge detection", &lowThreshold, 100, onTrackbar, nullptr);
        createTrackbar("High (%)", "Canny edge detection", &highThreshold, 100, onTrackbar, nullptr);

        while (cap.read(frame)) {
            canny(frame);
            if ((char)waitKey(30) == 'q') break;
        }
    }


    cap.release();
}

int main()
{
    // Config
    string inPath = "in.jpg",
        propPath = "config.properties";
    ifstream propFile(propPath);
    if (propFile.is_open()) {
        string line;
        while (getline(propFile, line)) {
            size_t equalsPos = line.find('=');
            if (equalsPos != string::npos) {
                string key = line.substr(0, equalsPos);
                string value = line.substr(equalsPos + 1);

                if (key == "headless") {
                    headless = (value == "true");
                }
                else if (key == "media") {
                    media = value;
                }
                else if (key == "in") {
                    inPath = value;
                }
                else if (key == "lowThreshold") {
                    lowThreshold = stoi(value);
                }
                else if (key == "highThreshold") {
                    highThreshold = stoi(value);
                }
            }
        }
    }
    else {
        printf("Could not open or find the properties file: %s\n", propPath.c_str());
    }

    initGauss();

    if (media == "image") {
        handleImage(inPath);
    }
    else {
        handleVideo(inPath);
    }

    hipFree(devGaussKernel);
    return 0;
}